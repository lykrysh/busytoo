#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>
using namespace std;

template <typename T>
struct square {
	__host__ __device__ T operator()(const T &x) const
	{
		return x * x;
	}
};

int main()
{
	float hostarr[4] = {1.0, 2.0, 3.0, 4.0}; // host array
	thrust::device_vector<float> dv(hostarr, hostarr + 4); // transfered to device

	square<float> unary_op;
	thrust::plus<float> binary_op;
	float init = 0;

	float norm = sqrt(thrust::transform_reduce(dv.begin(), dv.end(), unary_op, init, binary_op));
	cout << norm << endl;
	return 0;
}

