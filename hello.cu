
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;
#define N 4

__global__ void mykernel(void) {}

__global__ void add(int *a, int *b, int *sum) {
	int tid = blockIdx.x;
	if(tid < N)
		sum[tid] = a[tid] * b[tid];
}

int main() {
	mykernel<<<1,1>>>();

	int a[N],b[N],c[N];
	int *d_a, *d_b, *d_c;
	int sz = N * sizeof(int);

	hipMalloc((void **)&d_a, sz);
	hipMalloc((void **)&d_b, sz);
	hipMalloc((void **)&d_c, sz);

	for(int i=0; i<N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}

	hipMemcpy(d_a, a, sz, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sz, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, sz, hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);


	return 0;
}
