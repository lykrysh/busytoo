
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(uint n, float a, float *x, float *y) {
  uint i = blockIdx.x*blockDim.x + threadIdx.x; // nvcc built-ins
  if(i < n)
    y[i] = a*x[i] + y[i];
  }

void misc(void) {
  int ndev;
  hipDeviceProp_t prop;
  hipGetDeviceCount(&ndev);
  printf("This machine has %d CUDA devices.\n", ndev);
  for(int i = 0; i < ndev; i++) {
    const char *indent = (ndev == 0) ? "" : "  ";
    hipGetDeviceProperties(&prop, i);
    if(ndev > 0)
      printf("Device %d:\n", i);
    printf("%sdevice.name = %s\n", indent, prop.name);
    printf("%sdevice.maxThreadsPerBlock = %d\n", indent, prop.maxThreadsPerBlock);
  }
}

int main(int argc, char **argv) {
  uint n = atoi(argv[1]);
  int size = n*sizeof(float);
  float *x, *y, *yy; 
  float *dev_x, *dev_y;

  misc();
  x = (float *)malloc(size);
  y = (float *)malloc(size);
  yy = (float *)malloc(size);

  for(int i = 0; i < n; i++) {
    x[i] = i;
    y[i] = i*i;
  }

  hipMalloc((void**)(&dev_x), size);
  hipMalloc((void**)(&dev_y), size);
  hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, size, hipMemcpyHostToDevice);

  float a = 3.0;
  saxpy<<<ceil(n/256.0),256>>>(n, a, dev_x, dev_y);
  hipMemcpy(yy, dev_y, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < n; i++) { // check the result
    if(yy[i] != a*x[i] + y[i]) {
      fprintf(stderr, "ERROR: i=%d, a = %s, x[i]=%f, y[i]=%f, yy[i]=%f\n",
	      i, a, x[i], y[i], yy[i]);
      exit(-1);
    }
  }
  printf("The results match!\n");

  free(x);
  free(y);
  free(yy);
  hipFree(dev_x);
  hipFree(dev_y);
  exit(0);
}
