#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <list>
#include <vector>
#include <iostream>

using namespace std;

#define N 10

int main()
{
	thrust::device_vector<int> D(N, 1);
	thrust::fill(D.begin(), D.begin() + 7, 9); // 9 9 9 9 9 9 9 1 1 1

	thrust::host_vector<int> H(D.begin(), D.begin() + 5); // 9 9 9 9 9
	thrust::sequence(H.begin(), H.end()); // 0 1 2 3 4

	// copy all H to D
	thrust::copy(H.begin(), H.end(), D.begin());

	// can copy back to H
	H.resize(N);
	thrust::copy(D.begin(), D.end(), H.begin());

	cout << "H: ";
	thrust::copy(H.begin(), H.end(), ostream_iterator<int>(cout, " "));
	cout << endl;

	cout << "D is: ";
	thrust::copy(D.begin(), D.end(), ostream_iterator<int>(cout, " "));
	cout << endl;

	// OK to use regular vector too
	vector<int> regular(N);
	thrust::copy(D.begin(), D.end(), regular.begin());

	cout << "Regular vec is: ";
	thrust::copy(regular.begin(), regular.end(), ostream_iterator<int>(cout, " "));
	cout << endl;

	///////////////////////////////////////////////////////////////////
	// how to use raw pointer to device mem
	int *ptr;
	hipMalloc((void **) &ptr, N * sizeof(int));
	thrust::device_ptr<int> dev_ptr(ptr);
	//then we can do this
	thrust::fill(dev_ptr, dev_ptr + N, (int)99);

	cout << "Raw ptr is: ";
	for(auto i = dev_ptr; i != dev_ptr + N; ++i)
		cout << *i << " ";
	cout << endl;

	// how to extract raw pointer from device pointer (then what?)
	thrust::device_ptr<int> dev_ptr2 = thrust::device_malloc<int> (N);
	int *ptr2 = thrust::raw_pointer_cast(dev_ptr2);
	int *ptr3 = thrust::raw_pointer_cast(dev_ptr);

	// segfault below

	/*
	cout << "Casted ptr is: ";
	for(auto i = ptr2; i != ptr2 + N; ++i)
		cout << *i << " ";
	cout << endl;
	*/

	/////////////////////////////////////////////////////////////////
	// however using iterator (compared to pointer) is better choice for traversing
	list<int> lst;
	lst.push_back(10);
	lst.push_back(20);
	lst.push_back(30);
	lst.push_back(40);

	thrust::device_vector<int> DD(lst.begin(), lst.end());

	cout << "DD is: ";
	thrust::copy(DD.begin(), DD.end(), ostream_iterator<int>(cout, " "));
	cout << endl;

	return 0;
}
