#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <list>
#include <vector>

using namespace std;

#define N 10

int main()
{
	thrust::device_vector<int> D(N, 1);
	thrust::fill(D.begin(), D.begin() + 7, 9); // 9 9 9 9 9 9 9 1 1 1

	thrust::host_vector<int> H(D.begin(), D.begin() + 5); // 9 9 9 9 9
	thrust::sequence(H.begin(), H.end()); // 0 1 2 3 4

	// copy all H to D
	thrust::copy(H.begin(), H.end(), D.begin());

	// can copy back to H
	H.resize(N);
	thrust::copy(D.begin(), D.end(), H.begin());

	cout << "H is: ";
	for(auto i = H.begin(); i != H.end(); ++i)
		cout << *i << " ";
	cout << endl;

	cout << "D is: ";
	for(auto i = D.begin(); i != D.end(); ++i)
		cout << *i << " ";
	cout << endl;

	// OK to use regular vector too
	vector<int> regular(N);
	thrust::copy(D.begin(), D.end(), regular.begin());

	cout << "Regular vec is: ";
	for(auto i = regular.begin(); i != regular.end(); ++i)
		cout << *i << " ";
	cout << endl;

	///////////////////////////////////////////////////////////////////
	// how to use raw pointer to device mem
	int *ptr;
	hipMalloc((void **) &ptr, N * sizeof(int));
	thrust::device_ptr<int> dev_ptr(ptr);
	//then we can do this
	thrust::fill(dev_ptr, dev_ptr + N, (int)99);

	cout << "Raw ptr is: ";
	for(auto i = dev_ptr; i != dev_ptr + N; ++i)
		cout << *i << " ";
	cout << endl;

	// how to extract raw pointer from device pointer (then what?)
	thrust::device_ptr<int> dev_ptr2 = thrust::device_malloc<int> (N);
	int *ptr2 = thrust::raw_pointer_cast(dev_ptr2);
	int *ptr3 = thrust::raw_pointer_cast(dev_ptr);

	// segfault below

	/*
	cout << "Casted ptr is: ";
	for(auto i = ptr2; i != ptr2 + N; ++i)
		cout << *i << " ";
	cout << endl;
	*/

	/////////////////////////////////////////////////////////////////
	// however using iterator (compared to pointer) is better choice for traversing
	list<int> lst;
	lst.push_back(10);
	lst.push_back(20);
	lst.push_back(30);
	lst.push_back(40);

	thrust::device_vector<int> DD(lst.begin(), lst.end());

	cout << "DD is: ";
	for(auto i = DD.begin(); i != DD.end(); ++i)
		cout << *i << " ";
	cout << endl;

	return 0;
}
